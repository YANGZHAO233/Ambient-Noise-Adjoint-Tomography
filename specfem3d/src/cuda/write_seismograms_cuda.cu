#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                              CNRS, France
 !                       and Princeton University, USA
 !                 (there are currently many more authors!)
 !                           (c) October 2017
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 3 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
*/

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_elastic_seismogram_kernel(int nrec_local,
                                                  realw* field,
                                                  int* d_ibool,
                                                  realw* hxir, realw* hetar, realw* hgammar,
                                                  realw* seismograms,
                                                  realw* nu,
                                                  int* ispec_selected_rec_loc,
                                                  int it){

  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;

  // local index
  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  __shared__ realw sh_dxd[NGLL3_PADDED];
  __shared__ realw sh_dyd[NGLL3_PADDED];
  __shared__ realw sh_dzd[NGLL3_PADDED];

  if (irec_local < nrec_local) {

    int ispec = ispec_selected_rec_loc[irec_local] - 1;

    sh_dxd[tx] = 0;
    sh_dyd[tx] = 0;
    sh_dzd[tx] = 0;

    if (tx < NGLL3) {
      realw hlagrange = hxir[irec_local + nrec_local*I]*hetar[irec_local + nrec_local*J]*hgammar[irec_local + nrec_local*K];
      int iglob = iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,I,J,K,ispec)]-1;

      sh_dxd[tx] = hlagrange * field[0 + 3*iglob];
      sh_dyd[tx] = hlagrange * field[1 + 3*iglob];
      sh_dzd[tx] = hlagrange * field[2 + 3*iglob];

      //debug
      //if (tx == 0) printf("thread %d %d %d - %f %f %f\n",ispec,iglob,irec_local,hlagrange,field[0 + 2*iglob],field[1 + 2*iglob]);
    }
    __syncthreads();

    // reduction
    for (unsigned int s=1; s<NGLL3_PADDED ; s *= 2) {
      if (tx % (2*s) == 0){ sh_dxd[tx] += sh_dxd[tx + s];
                            sh_dyd[tx] += sh_dyd[tx + s];
                            sh_dzd[tx] += sh_dzd[tx + s];}
      __syncthreads();
    }

    int idx = INDEX3(NDIM,nrec_local,0,irec_local,it);

    if (tx == 0) {
      seismograms[0+idx] = nu[0+3*(0+3*irec_local)]*sh_dxd[0] + nu[0+3*(1+3*irec_local)]*sh_dyd[0] + nu[0+3*(2+3*irec_local)]*sh_dzd[0];
    }
    if (tx == 1) {
      seismograms[1+idx] = nu[1+3*(0+3*irec_local)]*sh_dxd[0] + nu[1+3*(1+3*irec_local)]*sh_dyd[0] + nu[1+3*(2+3*irec_local)]*sh_dzd[0];
    }
    if (tx == 2) {
      seismograms[2+idx] = nu[2+3*(0+3*irec_local)]*sh_dxd[0] + nu[2+3*(1+3*irec_local)]*sh_dyd[0] + nu[2+3*(2+3*irec_local)]*sh_dzd[0];
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_acoustic_seismogram_kernel(int nrec_local,
                                                   field* pressure,
                                                   int* d_ibool,
                                                   realw* hxir, realw* hetar, realw* hgammar,
                                                   field* seismograms,
                                                   int* ispec_selected_rec_loc,
                                                   int it){

  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;

  // local index
  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  __shared__ field sh_dxd[NGLL3_PADDED];

  if (irec_local < nrec_local) {

    int ispec = ispec_selected_rec_loc[irec_local]-1;

    sh_dxd[tx] = Make_field(0.f);

    if (tx < NGLL3) {

      realw hlagrange = hxir[irec_local + nrec_local*I]*hetar[irec_local + nrec_local*J]*hgammar[irec_local + nrec_local*K];
      int iglob = iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,I,J,K,ispec)]-1;

      sh_dxd[tx] = hlagrange*pressure[iglob];
    }
    __syncthreads();

    for (unsigned int s=1; s<NGLL3_PADDED ; s *= 2) {
      if (tx % (2*s) == 0) {sh_dxd[tx] += sh_dxd[tx + s];}
      __syncthreads();
    }

    int idx = INDEX2(nrec_local,irec_local,it);

    // Signe moins car pression = -potential_dot_dot
   if (tx == 0) seismograms[idx] = -sh_dxd[0];
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_acoustic_vectorial_seismogram_kernel(int nrec_local,
                   int*  d_ispec_is_acoustic,
                   field* scalar_potential,
                   realw* seismograms,
                   realw* d_rhostore,
                   int* d_ibool,
                   int * d_irregular_element_number,
                   realw* hxir, realw* hetar, realw* hgammar,
                   realw* d_xix, realw* d_xiy, realw* d_xiz,
                   realw* d_etax, realw* d_etay, realw* d_etaz,
                   realw* d_gammax, realw* d_gammay, realw* d_gammaz,
                   realw xix_regular,
                   realw* d_hprime_xx,
                   realw* nu,
                   int* ispec_selected_rec_loc,
                   int it){

  int irec_local = blockIdx.x + blockIdx.y*gridDim.x;
  int tx = threadIdx.x;

  // shared memory
  __shared__ realw s_dummy_loc[NGLL3_PADDED];
  __shared__ realw s_temp1[NGLL3_PADDED];
  __shared__ realw s_temp2[NGLL3_PADDED];
  __shared__ realw s_temp3[NGLL3_PADDED];
  __shared__ realw sh_hprime_xx[NGLL2];

  // locals
  realw temp1l, temp2l, temp3l;
  realw rho_invl, hlagrange;
  realw xixl, xiyl, xizl;
  realw etaxl, etayl, etazl;
  realw gammaxl, gammayl, gammazl;
  realw dpotentialdxl, dpotentialdyl, dpotentialdzl;
  int ispec, offset, offset_irreg, iglob, ispec_irreg;

  /*
  // debug
  if (irec_local < nrec_local) {
    ispec = ispec_selected_rec_loc[irec_local] - 1;
    offset = INDEX4_PADDED(NGLLX,NGLLX,NGLLX,I,J,K,ispec);
    iglob = d_ibool[offset]-1;
    rho_invl = 1.f / d_rhostore[offset];
    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    hlagrange = hxir[irec_local + nrec_local*I]*hetar[irec_local + nrec_local*J]*hgammar[irec_local + nrec_local*K];
    // loads into shared memory
    if (tx < NGLL2) {
      sh_hprime_xx[tx] = d_hprime_xx[tx];}
    s_dummy_loc[tx] = 1.; //scalar_potential[iglob];
    if (iglob > 0) {
      printf(" iglob =%d, (i,j,k)=(%d,%d,%d), ispec =%d  --- %f \n", iglob, I, J, K, ispec, scalar_potential[iglob]);}
    else{
      printf(" -illegal %d  %d %d %d %d\n", tx, ispec, I, J, K);
    }
  }
  */

  s_temp1[tx] = 0.0f;
  s_temp2[tx] = 0.0f;
  s_temp3[tx] = 0.0f;

  // local index
  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);


  if (irec_local >= nrec_local) return;

  if (tx < NGLL3) {
    ispec = ispec_selected_rec_loc[irec_local] - 1;
    ispec_irreg = d_irregular_element_number[ispec] - 1;
    // nothing to do if we are in elastic element
    if (d_ispec_is_acoustic[ispec] == 0) {return;}

    offset = INDEX4_PADDED(NGLLX,NGLLX,NGLLX,I,J,K,ispec);
    offset_irreg = INDEX4_PADDED(NGLLX,NGLLX,NGLLX,I,J,K,ispec_irreg);

    iglob = d_ibool[offset]-1;
    rho_invl = 1.f / d_rhostore[offset];
    hlagrange = hxir[irec_local + nrec_local*I]*hetar[irec_local + nrec_local*J]*hgammar[irec_local + nrec_local*K];
  }

  //debug
  //if (tx == 0) printf("thread %d %d %d - %f %f %f\n",ispec,iglob,irec_local,hlagrange,rho_invl, xixl);

  // loads into shared memory
  if (tx < NGLL2) sh_hprime_xx[tx] = d_hprime_xx[tx];
  if (tx < NGLL3) s_dummy_loc[tx] = (realw)scalar_potential[iglob];


  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready
  __syncthreads();

  if (tx < NGLL3) {
    // computes first matrix product
    temp1l = 0.f;
    temp2l = 0.f;
    temp3l = 0.f;

    for (int l=0;l<NGLLX;l++) {
      //assumes that hprime_xx = hprime_yy = hprime_zz
      // 1. cut-plane along xi-direction
      temp1l += s_dummy_loc[K*NGLL2+J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
      // 2. cut-plane along eta-direction
      temp2l += s_dummy_loc[K*NGLL2+l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
      // 3. cut-plane along gamma-direction
      temp3l += s_dummy_loc[l*NGLL2+J*NGLLX+I] * sh_hprime_xx[l*NGLLX+K];
    }

   if (ispec_irreg >= 0){ //irregular element
      xixl = d_xix[offset_irreg];
      xiyl = d_xiy[offset_irreg];
      xizl = d_xiz[offset_irreg];
      etaxl = d_etax[offset_irreg];
      etayl = d_etay[offset_irreg];
      etazl = d_etaz[offset_irreg];
      gammaxl = d_gammax[offset_irreg];
      gammayl = d_gammay[offset_irreg];
      gammazl = d_gammaz[offset_irreg];
      // compute derivatives of ux, uy and uz with respect to x, y and z
      // derivatives of potential
      dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
      dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
      dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;
    }
    else{
      // compute derivatives of ux, uy and uz with respect to x, y and z
      // derivatives of potential
      dpotentialdxl = xix_regular*temp1l;
      dpotentialdyl = xix_regular*temp2l;
      dpotentialdzl = xix_regular*temp3l;
    }

    // store the field in shared memmory
    s_temp1[tx] = hlagrange *dpotentialdxl * rho_invl;
    s_temp2[tx] = hlagrange *dpotentialdyl * rho_invl;
    s_temp3[tx] = hlagrange *dpotentialdzl * rho_invl;
  }

  __syncthreads();

  // reduction
  for (unsigned int s=1; s<NGLL3_PADDED ; s *= 2) {
    if (tx % (2*s) == 0){ s_temp1[tx] += s_temp1[tx + s];
                          s_temp2[tx] += s_temp2[tx + s];
                          s_temp3[tx] += s_temp3[tx + s];}
    __syncthreads();
  }

  int idx = INDEX3(NDIM,nrec_local,0,irec_local,it);

  if (tx == 0) {
    seismograms[0+idx] = nu[0+3*(0+3*irec_local)]*s_temp1[0] + nu[0+3*(1+3*irec_local)]*s_temp2[0] + nu[0+3*(2+3*irec_local)]*s_temp3[0];
  }
  if (tx == 1) {
    seismograms[1+idx] = nu[1+3*(0+3*irec_local)]*s_temp1[0] + nu[1+3*(1+3*irec_local)]*s_temp2[0] + nu[1+3*(2+3*irec_local)]*s_temp3[0];
  }
  if (tx == 2) {
    seismograms[2+idx] = nu[2+3*(0+3*irec_local)]*s_temp1[0] + nu[2+3*(1+3*irec_local)]*s_temp2[0] + nu[2+3*(2+3*irec_local)]*s_temp3[0];
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_seismograms_cuda,
              COMPUTE_SEISMOGRAMS_CUDA)(long* Mesh_pointer_f,
                                        realw* seismograms_d,
                                        realw* seismograms_v,
                                        realw* seismograms_a,
                                        realw* seismograms_p,
                                        int* seismo_currentf,
                                        int* NTSTEP_BETWEEN_OUTPUT_SEISMOSf,
                                        int* it, int* it_end,
                                        int* ACOUSTIC_SIMULATION,
                                        int* ELASTIC_SIMULATION,
                                        int* USE_TRICK_FOR_BETTER_PRESSURE) {

// compute_seismograms
  TRACE("compute_seismograms_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper

  //checks if anything to do
  if (mp->nrec_local == 0) return;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nrec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL3_PADDED,1,1);

  int seismo_current = *seismo_currentf - 1 ;
  int NTSTEP_BETWEEN_OUTPUT_SEISMOS = *NTSTEP_BETWEEN_OUTPUT_SEISMOSf;

  // warning: put in fortran routine prepare_GPU()
  /*
  if (it == 0){
    if (mp->save_seismograms_d || mp->save_seismograms_v || mp->save_seismograms_a){
      // warnings
      if (! *ELASTIC_SIMULATION)
        printf("\nWarning: Wrong type of seismogram for a pure fluid simulation, use pressure in seismotype\n");
      if (*ELASTIC_SIMULATION && *ACOUSTIC_SIMULATION)
        printf("\nWarning: Coupled elastic/fluid simulation has only valid displacement seismograms in elastic domain for GPU simulation\n\n");
    }
    if (mp->save_seismograms_p){
      if (! *ACOUSTIC_SIMULATION)
        printf("\nWarning: Wrong type of seismogram for a pure elastic simulation, use displ veloc or accel in seismotype\n");
      if (*ELASTIC_SIMULATION && *ACOUSTIC_SIMULATION)
        printf("\nWarning: Coupled elastic/fluid simulation has only valid pressure seismograms in fluid domain for GPU simulation\n\n");
    }
  }
  */


  // todo: for coupled simulations, one should check in which domain the receiver lies to output displacement
  //       similar to what routine compute_vector_one_element(..) is doing

  // computes current seismograms value

  // elastic wavefield
  // acoustic wavefield
  if (*ELASTIC_SIMULATION){
    if (mp->save_seismograms_d)
        compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                 mp->d_displ,
                                                                                 mp->d_ibool,
                                                                                 mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                 mp->d_seismograms_d,
                                                                                 mp->d_nu,
                                                                                 mp->d_ispec_selected_rec_loc,
                                                                                 seismo_current);


    if (mp->save_seismograms_v)
        compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                 mp->d_veloc,
                                                                                 mp->d_ibool,
                                                                                 mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                 mp->d_seismograms_v,
                                                                                 mp->d_nu,
                                                                                 mp->d_ispec_selected_rec_loc,
                                                                                 seismo_current);

    if (mp->save_seismograms_a)
        compute_elastic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                 mp->d_accel,
                                                                                 mp->d_ibool,
                                                                                 mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                 mp->d_seismograms_a,
                                                                                 mp->d_nu,
                                                                                 mp->d_ispec_selected_rec_loc,
                                                                                 seismo_current);

  }
  // acoustic wavefield
  if (*ACOUSTIC_SIMULATION){
    if (mp->save_seismograms_p){
        if (*USE_TRICK_FOR_BETTER_PRESSURE){
          compute_acoustic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                    mp->d_potential_acoustic,
                                                                                    mp->d_ibool,
                                                                                    mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                    mp->d_seismograms_p,
                                                                                    mp->d_ispec_selected_rec_loc,
                                                                                    seismo_current);
        }else{
          compute_acoustic_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                    mp->d_potential_dot_dot_acoustic,
                                                                                    mp->d_ibool,
                                                                                    mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                    mp->d_seismograms_p,
                                                                                    mp->d_ispec_selected_rec_loc,
                                                                                    seismo_current);
        }
    }

// VM VM add computation of vectorial field in fluids ----------------------------------------------------------------
    if (mp->save_seismograms_d)
      compute_acoustic_vectorial_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                          mp->d_ispec_is_acoustic,
                                                                                          mp->d_potential_acoustic,
                                                                                          mp->d_seismograms_d,
                                                                                          mp->d_rhostore,
                                                                                          mp->d_ibool,
                                                                                          mp->d_irregular_element_number,
                                                                                          mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                          mp->d_xix,mp->d_xiy,mp->d_xiz,
                                                                                          mp->d_etax,mp->d_etay,mp->d_etaz,
                                                                                          mp->d_gammax,mp->d_gammay,mp->d_gammaz,
                                                                                          mp->xix_regular,
                                                                                          mp->d_hprime_xx,
                                                                                          mp->d_nu,
                                                                                          mp->d_ispec_selected_rec_loc,
                                                                                          seismo_current);


    if (mp->save_seismograms_v)
      compute_acoustic_vectorial_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                          mp->d_ispec_is_acoustic,
                                                                                          mp->d_potential_dot_acoustic,
                                                                                          mp->d_seismograms_v,
                                                                                          mp->d_rhostore,
                                                                                          mp->d_ibool,
                                                                                          mp->d_irregular_element_number,
                                                                                          mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                          mp->d_xix,mp->d_xiy,mp->d_xiz,
                                                                                          mp->d_etax,mp->d_etay,mp->d_etaz,
                                                                                          mp->d_gammax,mp->d_gammay,mp->d_gammaz,
                                                                                          mp->xix_regular,
                                                                                          mp->d_hprime_xx,
                                                                                          mp->d_nu,
                                                                                          mp->d_ispec_selected_rec_loc,
                                                                                          seismo_current);


    if (mp->save_seismograms_a)
      compute_acoustic_vectorial_seismogram_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->nrec_local,
                                                                                          mp->d_ispec_is_acoustic,
                                                                                          mp->d_potential_dot_dot_acoustic,
                                                                                          mp->d_seismograms_a,
                                                                                          mp->d_rhostore,
                                                                                          mp->d_ibool,
                                                                                          mp->d_irregular_element_number,
                                                                                          mp->d_hxir,mp->d_hetar,mp->d_hgammar,
                                                                                          mp->d_xix,mp->d_xiy,mp->d_xiz,
                                                                                          mp->d_etax,mp->d_etay,mp->d_etaz,
                                                                                          mp->d_gammax,mp->d_gammay,mp->d_gammaz,
                                                                                          mp->xix_regular,
                                                                                          mp->d_hprime_xx,
                                                                                          mp->d_nu,
                                                                                          mp->d_ispec_selected_rec_loc,
                                                                                          seismo_current);
  } // ACOUSTIC_SIMULATION

  if (seismo_current == NTSTEP_BETWEEN_OUTPUT_SEISMOS || *it == *it_end ){
    int size = mp->nrec_local * NTSTEP_BETWEEN_OUTPUT_SEISMOS * sizeof(realw);

    // (hipMemcpy implicitly synchronizes all other cuda operations)
    if (mp->save_seismograms_d)
      print_CUDA_error_if_any(hipMemcpy(seismograms_d,mp->d_seismograms_d,NDIM * size,hipMemcpyDeviceToHost),72001);
    if (mp->save_seismograms_v)
      print_CUDA_error_if_any(hipMemcpy(seismograms_v,mp->d_seismograms_v,NDIM * size,hipMemcpyDeviceToHost),72002);
    if (mp->save_seismograms_a)
      print_CUDA_error_if_any(hipMemcpy(seismograms_a,mp->d_seismograms_a,NDIM * size,hipMemcpyDeviceToHost),72003);
    // EB EB Temporary solution : in the future we will also declare host pressure seismograms as (1,nrec_local,NTSTEP_BETWEEN_OUTPUT_SEISMOS)
    realw * seismo_temp;
    if (mp->save_seismograms_p){
      // EB EB We need to reorganize data to match host array shape :
      // if NB_RUNS_ACOUSTIC_GPU = 1 from fortran shape (1,nrec_local,NTSTEP_BETWEEN_OUTPUT_SEISMOS) to (NDIM,nrec_local,NTSTEP_BETWEEN_OUTPUT_SEISMOS)
      // if NB_RUNS_ACOUSTIC_GPU > 1 from fortran shape (NB_RUNS_ACOUSTIC_GPU,nrec_local,NTSTEP_BETWEEN_OUTPUT_SEISMOS) to (NDIM,nrec_local*NB_RUNS_ACOUSTIC_GPU,NTSTEP_BETWEEN_OUTPUT_SEISMOS)
      seismo_temp = (realw*)malloc(size*NB_RUNS_ACOUSTIC_GPU);
      print_CUDA_error_if_any(hipMemcpy(seismo_temp,mp->d_seismograms_p,size*NB_RUNS_ACOUSTIC_GPU,hipMemcpyDeviceToHost),72004);
      for (int it = 0; it<NTSTEP_BETWEEN_OUTPUT_SEISMOS; it++)
        for (int i_recloc=0; i_recloc<mp->nrec_local; i_recloc++)
          for (int i_run=0; i_run<NB_RUNS_ACOUSTIC_GPU; i_run++){
          seismograms_p[INDEX4(NDIM,mp->nrec_local,NB_RUNS_ACOUSTIC_GPU,0,i_recloc,i_run,it)] = seismo_temp[INDEX3(NB_RUNS_ACOUSTIC_GPU,mp->nrec_local,i_run,i_recloc,it)];
          seismograms_p[INDEX4(NDIM,mp->nrec_local,NB_RUNS_ACOUSTIC_GPU,1,i_recloc,i_run,it)] = 0.f;
          seismograms_p[INDEX4(NDIM,mp->nrec_local,NB_RUNS_ACOUSTIC_GPU,2,i_recloc,i_run,it)] = 0.f;
          }
      free(seismo_temp);
    }
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after compute_seismograms_cuda");
#endif
}

