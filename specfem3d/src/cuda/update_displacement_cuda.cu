#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                              CNRS, France
 !                       and Princeton University, USA
 !                 (there are currently many more authors!)
 !                           (c) October 2017
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 3 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// elastic wavefield

/* ----------------------------------------------------------------------------------------------- */


__global__ void UpdateDispVeloc_kernel(realw* displ,
                                       realw* veloc,
                                       realw* accel,
                                       int size,
                                       realw deltat,
                                       realw deltatsqover2,
                                       realw deltatover2) {

  // two dimensional array of blocks on grid where each block has one dimensional array of threads
  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  realw acc = accel[id];
  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    displ[id] = displ[id] + deltat*veloc[id] + deltatsqover2*acc;
    veloc[id] = veloc[id] + deltatover2*acc;
    accel[id] = 0.0f; // can do this using memset...not sure if faster,probably not
  }

// -----------------
// total of: 6 FLOP per thread (without int id calculation at beginning)
//
//           8 * 4 BYTE = 32 DRAM accesses per thread
//
// arithmetic intensity: 6 FLOP / 32 BYTES ~ 0.19 FLOP/BYTE
// -----------------
// nvprof: 24599250 flops for 4099875 threads -> 6 FLOP per thread
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(update_displacement_cuda,
              UPDATE_DISPLACMENT_CUDA)(long* Mesh_pointer,
                                          realw* deltat_F,
                                          realw* deltatsqover2_F,
                                          realw* deltatover2_F,
                                          realw* b_deltat_F,
                                          realw* b_deltatsqover2_F,
                                          realw* b_deltatover2_F) {

  TRACE("\tupdate_displacement_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  realw deltat = *deltat_F;
  realw deltatsqover2 = *deltatsqover2_F;
  realw deltatover2 = *deltatover2_F;

  int size = NDIM * mp->NGLOB_AB;

  int blocksize = BLOCKSIZE_KERNEL1;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // Cuda timing
  hipEvent_t start,stop;
  if (CUDA_TIMING_UPDATE ){
    start_timing_cuda(&start,&stop);
  }

  // debug
  //realw max_d,max_v,max_a;
  //max_d = get_device_array_maximum_value(mp->d_displ, size);
  //max_v = get_device_array_maximum_value(mp->d_veloc, size);
  //max_a = get_device_array_maximum_value(mp->d_accel, size);
  //printf("rank %d - max displ: %f veloc: %f accel: %f\n",mp->myrank,max_d,max_v,max_a);

  //launch kernel
  UpdateDispVeloc_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_displ,mp->d_veloc,mp->d_accel,
                                                                size,deltat,deltatsqover2,deltatover2);

  // kernel for backward fields
  if (mp->simulation_type == 3) {
    realw b_deltat = *b_deltat_F;
    realw b_deltatsqover2 = *b_deltatsqover2_F;
    realw b_deltatover2 = *b_deltatover2_F;

    UpdateDispVeloc_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_displ,mp->d_b_veloc,mp->d_b_accel,
                                                                  size,b_deltat,b_deltatsqover2,b_deltatover2);
  }

  // Cuda timing
  if (CUDA_TIMING_UPDATE ){
    realw flops,time;
    stop_timing_cuda(&start,&stop,"UpdateDispVeloc_kernel",&time);
    // time in seconds
    time = time / 1000.;
    // performance: 6 FLOPS per thread
    flops = 6.0 * size;
    //printf("  performance: %f GFlop/s num_blocks x/y: %d %d threads: %d\n", flops/time * 1.e-9,num_blocks_x,num_blocks_y,size);
    printf("  performance: %f GFlop/s\n", flops/time * 1.e-9);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("update_displacement_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// acoustic wavefield

// KERNEL 1
/* ----------------------------------------------------------------------------------------------- */

__global__ void UpdatePotential_kernel(field* potential_acoustic,
                                       field* potential_dot_acoustic,
                                       field* potential_dot_dot_acoustic,
                                       int size,
                                       realw deltat,
                                       realw deltatsqover2,
                                       realw deltatover2) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;

  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    field p_dot_dot = potential_dot_dot_acoustic[id];
    field p_dot = potential_dot_acoustic[id];
    potential_acoustic[id] +=   deltat*p_dot
                              + deltatsqover2*p_dot_dot;

    potential_dot_acoustic[id] = p_dot + deltatover2*p_dot_dot;

    potential_dot_dot_acoustic[id] = Make_field(0.f);
  }

// -----------------
// total of: 6 FLOP per thread (without id calculation)
//
//           8 * 4 BYTE = 32 DRAM accesses per thread
//
// arithmetic intensity: 6 FLOP / 32 BYTES ~ 0.19 FLOP/BYTE
// -----------------
//
// nvprof: nvprof --metrics flops_sp ./xspecfem3D
//          -> 8199750 FLOPS (Single) floating-point operations for 1366625 threads
//                                    1366625 (NGLOB) -> 10677 * 128 active threads- 31 ghost threads
//          -> 6 FLOP per thread


}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(it_update_displacement_ac_cuda,
              it_update_displacement_ac_cuda)(long* Mesh_pointer,
                                               realw* deltat_F,
                                               realw* deltatsqover2_F,
                                               realw* deltatover2_F,
                                               realw* b_deltat_F,
                                               realw* b_deltatsqover2_F,
                                               realw* b_deltatover2_F) {
  TRACE("\tit_update_displacement_ac_cuda");
  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  int size = mp->NGLOB_AB;

  int blocksize = BLOCKSIZE_KERNEL1;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  //launch kernel
  // forward wavefields
  realw deltat = *deltat_F;
  realw deltatsqover2 = *deltatsqover2_F;
  realw deltatover2 = *deltatover2_F;

  // Cuda timing
  hipEvent_t start,stop;
  if (CUDA_TIMING_UPDATE ){
    start_timing_cuda(&start,&stop);
  }

  UpdatePotential_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_acoustic,
                                                                 mp->d_potential_dot_acoustic,
                                                                 mp->d_potential_dot_dot_acoustic,
                                                                 size,deltat,deltatsqover2,deltatover2);

  // backward/reconstructed wavefields
  if (mp->simulation_type == 3) {
    realw b_deltat = *b_deltat_F;
    realw b_deltatsqover2 = *b_deltatsqover2_F;
    realw b_deltatover2 = *b_deltatover2_F;

    UpdatePotential_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_acoustic,
                                                                  mp->d_b_potential_dot_acoustic,
                                                                  mp->d_b_potential_dot_dot_acoustic,
                                                                  size,b_deltat,b_deltatsqover2,b_deltatover2);
  }

  // Cuda timing
  if (CUDA_TIMING_UPDATE ){
    realw flops,time;
    stop_timing_cuda(&start,&stop,"UpdatePotential_kernel",&time);
    // time in seconds
    time = time / 1000.;
    // performance
    // see with: nvprof --metrics flops_sp ./xspecfem3D
    //           -> using 8199750 FLOPS (Single) floating-point operations for 1366625 threads
    //              = 6 FLOPS per thread
    flops = 6.0 * size;
    //printf("  performance: %f GFlop/s num_blocks x/y: %d %d threads: %d\n", flops/time * 1.e-9,num_blocks_x,num_blocks_y,size);
    printf("  performance: %f GFlop/s\n", flops/time * 1.e-9);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("checking updatedispl_kernel launch...with %dx%d blocks\n",num_blocks_x,num_blocks_y);
  exit_on_cuda_error("it_update_displacement_ac_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// elastic domains

// KERNEL 3

/* ----------------------------------------------------------------------------------------------- */

__global__ void kernel_3_cuda_device(realw* veloc,
                                     realw* accel,
                                     realw* b_veloc,
                                     realw* b_accel,
                                     int size,
                                     int simulation_type,
                                     realw deltatover2,
                                     realw b_deltatover2,
                                     realw* rmassx,
                                     realw* rmassy,
                                     realw* rmassz) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  realw rx,ry,rz;
  realw ax,ay,az;
  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    rx = rmassx[id];
    ry = rmassy[id];
    rz = rmassz[id];
    ax = accel[3*id  ]*rx;
    ay = accel[3*id+1]*ry;
    az = accel[3*id+2]*rz;

    accel[3*id]   = ax;
    accel[3*id+1] = ay;
    accel[3*id+2] = az;

    veloc[3*id]   += deltatover2*ax;
    veloc[3*id+1] += deltatover2*ay;
    veloc[3*id+2] += deltatover2*az;

    if (simulation_type==3){
      ax = b_accel[3*id  ]*rx;
      ay = b_accel[3*id+1]*ry;
      az = b_accel[3*id+2]*rz;

      b_accel[3*id]   = ax;
      b_accel[3*id+1] = ay;
      b_accel[3*id+2] = az;

      b_veloc[3*id]   += b_deltatover2*ax;
      b_veloc[3*id+1] += b_deltatover2*ay;
      b_veloc[3*id+2] += b_deltatover2*az;

    }

  }

}

/* ----------------------------------------------------------------------------------------------- */

__global__ void kernel_3_accel_cuda_device(realw* accel,
                                           realw* b_accel,
                                           int size,
                                           int simulation_type,
                                           realw* rmassx,
                                           realw* rmassy,
                                           realw* rmassz) {
  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;

  realw rx,ry,rz;
  realw ax,ay,az;
  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    rx = rmassx[id];
    ry = rmassy[id];
    rz = rmassz[id];
    ax = accel[3*id  ]*rx;
    ay = accel[3*id+1]*ry;
    az = accel[3*id+2]*rz;
    accel[3*id  ] = ax;
    accel[3*id+1] = ay;
    accel[3*id+2] = az;

    if (simulation_type==3){
      ax = b_accel[3*id  ]*rx;
      ay = b_accel[3*id+1]*ry;
      az = b_accel[3*id+2]*rz;

      b_accel[3*id]   = ax;
      b_accel[3*id+1] = ay;
      b_accel[3*id+2] = az;

    }

  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void kernel_3_veloc_cuda_device(realw* veloc,
                                           realw* accel,
                                           int size,
                                           realw deltatover2) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;

  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    veloc[3*id] = veloc[3*id] + deltatover2*accel[3*id];
    veloc[3*id+1] = veloc[3*id+1] + deltatover2*accel[3*id+1];
    veloc[3*id+2] = veloc[3*id+2] + deltatover2*accel[3*id+2];
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(kernel_3_a_cuda,
              KERNEL_3_A_CUDA)(long* Mesh_pointer,
                               realw* deltatover2_F,
                               realw* b_deltatover2_F,
                               int* APPROXIMATE_OCEAN_LOAD) {

  TRACE("\tkernel_3_a_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  int size = mp->NGLOB_AB;

  int blocksize = BLOCKSIZE_KERNEL3;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // check whether we can update accel and veloc, or only accel at this point
  if (*APPROXIMATE_OCEAN_LOAD == 0){
   realw deltatover2 = *deltatover2_F;
   realw b_deltatover2 = *b_deltatover2_F;
   // updates both, accel and veloc
   kernel_3_cuda_device<<< grid, threads,0,mp->compute_stream>>>(mp->d_veloc,
                                                                 mp->d_accel,
                                                                 mp->d_b_veloc,
                                                                 mp->d_b_accel,
                                                                 size,mp->simulation_type,deltatover2,b_deltatover2,
                                                                 mp->d_rmassx,mp->d_rmassy,mp->d_rmassz);
  }else{
   // updates only accel
   kernel_3_accel_cuda_device<<< grid, threads,0,mp->compute_stream>>>(mp->d_accel,
                                                                       mp->d_b_accel,
                                                                       size,
                                                                       mp->simulation_type,
                                                                       mp->d_rmassx,
                                                                       mp->d_rmassy,
                                                                       mp->d_rmassz);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("checking updatedispl_kernel launch...with %dx%d blocks\n",num_blocks_x,num_blocks_y);
  exit_on_cuda_error("after kernel 3 a");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(kernel_3_b_cuda,
              KERNEL_3_B_CUDA)(long* Mesh_pointer,
                               realw* deltatover2_F,
                               realw* b_deltatover2_F) {
  TRACE("\tkernel_3_b_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  int size = mp->NGLOB_AB;

  int blocksize = BLOCKSIZE_KERNEL3;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  realw deltatover2 = *deltatover2_F;
  // updates only veloc at this point
  kernel_3_veloc_cuda_device<<< grid, threads,0,mp->compute_stream>>>(mp->d_veloc,
                                                                      mp->d_accel,
                                                                      size,deltatover2);

  if (mp->simulation_type == 3) {
    realw b_deltatover2 = *b_deltatover2_F;
    kernel_3_veloc_cuda_device<<< grid, threads,0,mp->compute_stream>>>(mp->d_b_veloc,
                                                                        mp->d_b_accel,
                                                                        size,b_deltatover2);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("checking updatedispl_kernel launch...with %dx%d blocks\n",num_blocks_x,num_blocks_y);
  exit_on_cuda_error("after kernel 3 b");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// acoustic domains

// KERNEL 3

/* ----------------------------------------------------------------------------------------------- */


__global__ void kernel_3_acoustic_cuda_device(field* potential_dot_acoustic,
                                                field* potential_dot_dot_acoustic,
                                                field* b_potential_dot_acoustic,
                                                field* b_potential_dot_dot_acoustic,
                                                int simulation_type,
                                                int size,
                                                realw deltatover2,
                                                realw b_deltatover2,
                                                realw* rmass_acoustic) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  realw rmass;
  field p_dot_dot;
  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    rmass = rmass_acoustic[id];
    // multiplies pressure with the inverse of the mass matrix
    p_dot_dot = rmass*potential_dot_dot_acoustic[id];
    potential_dot_dot_acoustic[id] = p_dot_dot;
    potential_dot_acoustic[id] += deltatover2*p_dot_dot;
    if (simulation_type==3) {
      p_dot_dot = rmass*b_potential_dot_dot_acoustic[id];
      b_potential_dot_dot_acoustic[id] = p_dot_dot;
      b_potential_dot_acoustic[id] += b_deltatover2*p_dot_dot;
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(kernel_3_acoustic_cuda,
              KERNEL_3_ACOUSTIC_CUDA)(long* Mesh_pointer,
                                      realw* deltatover2_F,
                                      realw* b_deltatover2_F) {

TRACE("kernel_3_acoustic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  int size = mp->NGLOB_AB;

  int blocksize = BLOCKSIZE_KERNEL3;
  int size_padded = ((int)ceil(((double)size)/((double)blocksize)))*blocksize;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(size_padded/blocksize,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  realw deltaover2 = *deltatover2_F;
  realw b_deltaover2 = *b_deltatover2_F;

  kernel_3_acoustic_cuda_device<<< grid, threads>>>(mp->d_potential_dot_acoustic,
                                                    mp->d_potential_dot_dot_acoustic,
                                                    mp->d_b_potential_dot_acoustic,
                                                    mp->d_b_potential_dot_dot_acoustic,
                                                    mp->simulation_type,
                                                    size,
                                                    deltaover2,
                                                    b_deltaover2,
                                                    mp->d_rmass_acoustic);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("checking updatedispl_kernel launch...with %dx%d blocks\n",num_blocks_x,num_blocks_y);
  exit_on_cuda_error("after kernel 3 ");
#endif
}

